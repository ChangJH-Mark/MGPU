
#include <hip/hip_runtime.h>
/* ==========Original============*/
extern "C" __global__ void
vectorAdd(const int *A, const int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/*====================mgpu====================*/
#define SMID_MASK 0xf
#define ITERS_MASK 0xf0
#define WORKER_MASK 0xff00
#define ITERS 10
#define MAX_SM 6
#define GET_SID(flags) (flags & SMID_MASK)
#define GET_WID(flags) (( flags & WORKER_MASK) >> 8)
#define SET_WID(flags, worker) (flags = (flags & ~(WORKER_MASK)) + ((worker) << 8))
#define GET_ITERS(flags) ((flags & ITERS_MASK) >> 4)
#define SET_ITERS(flags, times) (flags = (flags & ~(ITERS_MASK)) + ((times) << 4))
#define IS_LEAD_THREAD (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)

// __device__ int sms_flag;// block limits per sm - max sm id - min sm id
// __device__ int b_cnts; // total block counts
// __device__ int b_fins; // finished block counts
// __device__ dim3 gridDIM; // origin grid dim
// __device__ int w_cnts[MAX_SM]; // workers per sm
__device__ int configs[6 + MAX_SM]; // 0: sms_flag; 1: b_cnts; 2: b_fins; 3~5 : gridDIM; 6~end w_cnts[MAX_SM]
#define SMS_FLAG (configs[0])
#define TOTAL_CNTS (configs[1])
#define FIN_CNTS (configs[2])
#define GridDim_X (configs[3])
#define GridDim_Y (configs[4])
#define GridDim_Z (configs[5])
#define WORKER_ADDR (configs + 6)
#define GET_SM_LOW(sms_flag) ((sms_flag) & 0xff)
#define GET_SM_HIGH(sms_flag) (((sms_flag) & 0xff00) >> 8)
#define GET_BLOCK_LIMIT(sms_flag) (((sms_flag) & 0xffff0000) >> 16)

__device__ void vectorAdd_V(const int *A, const int *B, int *C, int numElements, uint3 blockIDX);

extern "C" __global__ void vectorAdd_V1(const int *A, const int *B, int *C, int numElements){
    __shared__ int start_block;
    __shared__ int flags;
    // set sid & wid
    if(IS_LEAD_THREAD) {
        int worker;
        start_block = 0;
        flags = 0;
        asm("mov.u32 %0, %smid;":"=r"(flags));
        // sm check
        if(GET_SID(flags) < GET_SM_LOW(SMS_FLAG) || GET_SID(flags) > GET_SM_HIGH(SMS_FLAG)) {
            start_block = -1;
        } else if((worker = atomicAdd(WORKER_ADDR + GET_SID(flags), 1)) >= GET_BLOCK_LIMIT(SMS_FLAG)) {
            start_block = -1;
        } else {
            SET_WID(flags, worker);
        }
    }
    __syncthreads();
    if(start_block == -1)
        goto end;
    __shared__ uint3 blockIDX;

    while(start_block != -1) {
        if(IS_LEAD_THREAD) {
            // sm check
            if(GET_SID(flags) < GET_SM_LOW(SMS_FLAG) || GET_SID(SMS_FLAG) > GET_SM_HIGH(SMS_FLAG)) {
                start_block = -1;
            } else if(GET_WID(flags) >= GET_BLOCK_LIMIT(SMS_FLAG)) {
                // worker check
                start_block = -1;
            } else {
                start_block = atomicAdd(&FIN_CNTS, ITERS);
                if(start_block >= TOTAL_CNTS)
                    start_block = -1;
                else {
                    SET_ITERS(flags, min(start_block + ITERS, TOTAL_CNTS) - start_block);
                    blockIDX = make_uint3(start_block % GridDim_X, (start_block / GridDim_X) % GridDim_Y, (start_block / (GridDim_X * GridDim_Y)));
                }
            }
        }// if threadIdx.x == 0
        __syncthreads();
        if(start_block == -1) {
            goto end;
        }
#pragma unroll
        for(int i = start_block; i < start_block + GET_ITERS(flags);) {
            vectorAdd_V(A, B, C, numElements, blockIDX);
            i++;
            if(ITERS > 1 && (IS_LEAD_THREAD)) {
                blockIDX = make_uint3(i % GridDim_X, (i / GridDim_X) % GridDim_Y, (i / (GridDim_X * GridDim_Y)));
            }
            __syncthreads();
        }
    }// while
    end:
    if(IS_LEAD_THREAD)
        atomicSub(WORKER_ADDR + GET_SID(flags), 1);
}

__device__ void
vectorAdd_V(const int *A, const int *B, int *C, int numElements, uint3 blockIDX)
{
    int i = blockDim.x * blockIDX.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}