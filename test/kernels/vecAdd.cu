
#include <hip/hip_runtime.h>
//
// Created by root on 2021/3/25.
//
#define ITERS 10
__device__ void vecAdd(int *a, int *b, int num, uint3 blockIDX, dim3 gridDIM);

__device__ uint get_smid() {
    uint ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret));
    return ret;
}

__device__ int finished = 0;

extern "C" __global__ void vecAddProxy(int *a, int *b, int num, int sm_low, int sm_high, dim3 grid, int blocks)
{
    // reside on sm (sm >= sm_low && sm < sm_high)
    bool leader = false;
    __shared__ bool terminate;
    if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        leader = true;
    }
    if(leader)
    {
        terminate = false;
        int sm_id = get_smid();
        if(sm_id < sm_low || sm_id >= sm_high) {
            terminate = true;
        }
    }
    __syncthreads();
    if(terminate)
        return;
    // do jobs iterately
    __shared__ int index;
    index = 0;
    while(index < blocks)
    {
        // detect if finished blocks over boundary
        if(leader)
        {
            index = atomicAdd(&finished, ITERS);
            if(index >= blocks) {
                terminate = true;
            }
        }
        __syncthreads();
        if(terminate)
            return;
        int high_boundary = min(index + ITERS, blocks);
        for(int i = index; i < high_boundary; i++)
        {
            uint3 blockIDX = make_uint3( i % grid.x, (i / grid.x) % grid.y, (i / (grid.x * grid.y)));
            vecAdd(a, b, num, blockIDX, grid);
        }
    }
}

__device__ void vecAdd(int *a, int *b, int num, uint3 blockIDX, dim3 gridDIM) {
    int skip = gridDIM.x * blockDim.x;
    for(int i= threadIdx.x + blockIDX.x * blockDim.x; i< num; i+= skip){
        b[i] = a[i] + b[i];
    }
}

extern "C" __global__ void vecAdd(int *a, int *b, int num) {
    int skip = gridDim.x * blockDim.x;
    for(int i = threadIdx.x + blockIdx.x * blockDim.x; i < num; i+= skip) {
        b[i] = a[i] + b[i];
    }
}